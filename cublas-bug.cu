
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <string>
#include <iostream>

#include "hipblas.h"


/*
 * Host code
 */
int main(int argc, char *argv[])
{
    // Compute C = transpose(A)*B for large A (>2^31 elements) using cuBLAS

    hipblasHandle_t cublas_h;

    printf("Init cuBLAS...\n");
    hipblasCreate(&cublas_h);

    float* A;
    float* B;
    float* C;

    // Size of main matrix A
    size_t Ar = 24000; // rows of A
    size_t Ac = 100000; // columns of A

    size_t Bc = 3; // Columns of B

    size_t N_A = Ar*Ac; // bug is triggered if Bc > 1 and N_A > 2^31
    size_t N_B = Ar*Bc;
    size_t N_C = Ac*Bc;

    hipMallocManaged(&A, sizeof(float)*N_A);
    hipMallocManaged(&B, sizeof(float)*N_B);
    hipMallocManaged(&C, sizeof(float)*N_C);

    printf("N_A = %zi (%g)\n", N_A, (double)N_A);

    // Set values in A and B
    printf("Setting values for A and B\n");
    for(size_t i = 0; i < N_A; i++)
	A[i] = 1.0f;

    for(size_t i = 0; i < N_B; i++)
	B[i] = 1.0f;

    // Call cuBLAS - alpha*op(A)*op(B) + beta*C
    const size_t m = Ac;  // rows of op(A) and C
    const size_t n = Bc;   // columns of op(B) and C
    const size_t k = Ar; // columns op op(A) and rows of op(B)
    float alpha = 1.0f;
    float beta = 0.0f;

    size_t lda = Ar;
    size_t ldb = Ar;
    size_t ldc = Ac;

    printf("Call cuBLAS\n");
    hipblasStatus_t errn =
	hipblasSgemm(cublas_h, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k,
		&alpha, A, lda, B, ldb, &beta, C, ldc);

    if(errn == HIPBLAS_STATUS_SUCCESS)
	printf("cuBLAS success\n");
    else
	printf("cuBLAS error\n");

    printf("cudaDeviceSynchronize()\n");
    hipDeviceSynchronize();

    printf("C[0] = %g\n", C[0]); // This is correct == Ar
    printf("C[%zi] = %g\n", N_C-1, C[N_C-1]); // This is incorrect when Bc > 1 and N_A > 2^31

    hipFree(A);
    hipFree(B);
    hipFree(C);

}
